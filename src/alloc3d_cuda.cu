#include <stdlib.h>
#include <hip/hip_runtime.h>

double *malloc_3d_cuda(int m, int n, int k) {

    if (m <= 0 || n <= 0 || k <= 0)
        return NULL;

    // Calculate the total number of elements in the 3D matrix
    const int numElements = m * n * k;

    // Calculate the size of memory required in bytes
    const int sizeBytes = numElements * sizeof(double);

    // Allocate memory on the device (GPU)
    double *d_matrix;
    hipMalloc((void**)&d_matrix, sizeBytes);

    // Check if the memory allocation was successful
    if (d_matrix == nullptr) {
        return NULL;
    }

    return d_matrix;

}

void free_3d_cuda(double *array3D) {
    hipFree(array3D);
}

